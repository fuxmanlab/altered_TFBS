
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define PROMOTER_LEN 51594
#define SCORE_LEN 1963
#define PNVEC_MAX 18000

#define Nchunk 5000
#define MAX_LINE_LEN 24000
#define DELIM "\t"


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__global__ 
void tfbs_kernel(int N,    int pvec_max,
                 char *pvec,   int *pvec_len, 
                 float *pwm,    int pwm_len, 
                 float tf_threshold,
                 float *bg,

                 float *output) 

{

   int id = blockIdx.x * blockDim.x + threadIdx.x;
   int ind1, ind2;
   float wt_score1, wt_score2;
   float wt_score31, wt_score41;
   float wt_score32, wt_score42;
   float wt_score33, wt_score43;
   float wt_score34, wt_score44;
   int ioffset = 29; 


   if (id < N ) {

        int ncol = 4;


      /* loop through positions */ 
      for ( int ipos = ioffset; ipos < pvec_len[id] - ioffset; ipos++ ){



         float fs1 = -99999.;
         float fs2 = -99999.;
         float fs31 = -99999.;
         float fs41 = -99999.;
         float fs32 = -99999.;
         float fs42 = -99999.;
         float fs33 = -99999.;
         float fs43 = -99999.;
         float fs34 = -99999.;
         float fs44 = -99999.;
         
         /* loop through all k-mers for that position */
         for (   int ik = 0; ik < pwm_len; ik ++ ) {
            
          
            wt_score1 = 0; wt_score2 = 0; 
	    wt_score31 = 0; wt_score41=0;
	    wt_score32 = 0; wt_score42=0;
	    wt_score33 = 0; wt_score43=0;
	    wt_score34 = 0; wt_score44=0;

            /* loop through the k-mer */
            for (   int i = 0; i < pwm_len; i++){


               ind1 = pvec[id * pvec_max + ipos + ik - pwm_len + 1 + i];
               if ( ind1 < 1 ) { /* skip kmere if not A,C,T,G */
                                        wt_score1 = 0; wt_score2 = 0;
                                        wt_score31 = 0; wt_score41=0;
                                        wt_score32 = 0; wt_score42=0;
                                        wt_score33 = 0; wt_score43=0;
                                        wt_score34 = 0; wt_score44=0;
                                        break;
                                }

               ind2 = 5 - pvec[id * pvec_max + ipos + ik   - i ];
               if ( ind2 > 4 ) { /* skip kmere if not A,C,T,G */
                                        wt_score1 = 0; wt_score2 = 0;
                                        wt_score31 = 0; wt_score41=0;
                                        wt_score32 = 0; wt_score42=0;
                                        wt_score33 = 0; wt_score43=0;
                                        wt_score34 = 0; wt_score44=0;
                                        break;
                                }


               wt_score1 = wt_score1 + log(pwm[ (i * ncol )  +  (ind1 -1) ] / bg[ ind1 - 1 ] );
               wt_score2 = wt_score2 + log(pwm[ (i * ncol )  +  (ind2 -1) ] / bg[ ind2 - 1 ] );

               if ( i == (pwm_len -ik -1)) ind1 = 1;
               if ( i == ik) ind2 = 5 - 1;
               wt_score31 = wt_score31 + log(pwm[ (i * ncol )  +  (ind1 -1) ] / bg[ ind1 - 1 ] );
               wt_score41 = wt_score41 + log(pwm[ (i * ncol )  +  (ind2 -1) ] / bg[ ind2 - 1 ] );

               if ( i == (pwm_len -ik -1)) ind1 = 2;
               if ( i == ik) ind2 = 5 - 2;
               wt_score32 = wt_score32 + log(pwm[ (i * ncol )  +  (ind1 -1) ] / bg[ ind1 - 1 ] );
               wt_score42 = wt_score42 + log(pwm[ (i * ncol )  +  (ind2 -1) ] / bg[ ind2 - 1 ] );

               if ( i == (pwm_len -ik -1)) ind1 = 3;
               if ( i == ik) ind2 = 5 - 3;
               wt_score33 = wt_score33 + log(pwm[ (i * ncol )  +  (ind1 -1) ] / bg[ ind1 - 1 ] );
               wt_score43 = wt_score43 + log(pwm[ (i * ncol )  +  (ind2 -1) ] / bg[ ind2 - 1 ] );

               if ( i == (pwm_len -ik -1)) ind1 = 4;
               if ( i == ik) ind2 = 5 - 4;
               wt_score34 = wt_score34 + log(pwm[ (i * ncol )  +  (ind1 -1) ] / bg[ ind1 - 1 ] );
               wt_score44 = wt_score44 + log(pwm[ (i * ncol )  +  (ind2 -1) ] / bg[ ind2 - 1 ] );
            }

            /* for all kmers for this position calculate maximum scores */
            fs1 = ( fs1 < wt_score1 ) ? wt_score1 : fs1; 
            fs2 = ( fs2 < wt_score2 ) ? wt_score2 : fs2; 
            fs31 = ( fs31 < wt_score31 ) ? wt_score31 : fs31; 
            fs41 = ( fs41 < wt_score41 ) ? wt_score41 : fs41;
            fs32 = ( fs32 < wt_score32 ) ? wt_score32 : fs32; 
            fs42 = ( fs42 < wt_score42 ) ? wt_score42 : fs42;
            fs33 = ( fs33 < wt_score33 ) ? wt_score33 : fs33; 
            fs43 = ( fs43 < wt_score43 ) ? wt_score43 : fs43;
            fs34 = ( fs34 < wt_score34 ) ? wt_score34 : fs34; 
            fs44 = ( fs44 < wt_score44 ) ? wt_score44 : fs44;

         }

         /* if all the scores are less than threshold do not store them */
         int icol =  id * (pvec_max - ioffset * 2 )  +  (ipos - ioffset );
         int ichunk = N * (pvec_max - ioffset * 2 );
         output[ 1 * ichunk + icol ] = ipos + 1;

         output[ 2 * ichunk + icol ] = fs1;
         output[ 3  * ichunk + icol ] = fs2;


         if ( (fs31 >= tf_threshold | fs41 >= tf_threshold ) & pvec[ id * pvec_max + ipos] != 1 ){
            output[ icol ] = 1;
            output[ 4 * ichunk + icol ] = fs31;
            output[ 5 * ichunk + icol ] = fs41;
         } else if( ( fs1 > tf_threshold | fs2 > tf_threshold) &   pvec[ id * pvec_max + ipos] != 1 ){
            output[ icol ] =0;
            output[ 4 * ichunk + icol ] = fs31;
            output[ 5 * ichunk + icol ] = fs41;

         } else {
            output[ icol ]  = 0;

         }

         if ( ( fs32 >= tf_threshold | fs42 >= tf_threshold) & pvec[ id * pvec_max + ipos] != 2 ){
            output[ icol ] = output[  icol  ] + 10;
            output[ 6 * ichunk + icol ] = fs32;
            output[ 7 * ichunk + icol ] = fs42;
         } else if( ( fs1 > tf_threshold | fs2 > tf_threshold) &   pvec[ id * pvec_max + ipos] != 2 ){
            output[ 6 * ichunk + icol ] = fs32;
            output[ 7 * ichunk + icol ] = fs42;
         } 

         if ( ( fs33 >= tf_threshold | fs43 >= tf_threshold) & pvec[ id * pvec_max + ipos] != 3 ){
            output[ icol ] = output[  icol  ] + 100;
            output[ 8 * ichunk + icol ] = fs33;
            output[ 9 * ichunk + icol ] = fs43;
         } else if( ( fs1 > tf_threshold | fs2 > tf_threshold) &   pvec[ id * pvec_max + ipos] != 3 ){
            output[ 8 * ichunk + icol ] = fs33;
            output[ 9 * ichunk + icol ] = fs43;
         } 

         if ( ( fs34 >= tf_threshold | fs44 >= tf_threshold) & pvec[ id * pvec_max + ipos] != 4 ){
            output[ icol ] = output[ icol ] + 1000;
            output[ 10 * ichunk + icol ] = fs34;
            output[ 11 * ichunk + icol ] = fs44;
         } else if( ( fs1 > tf_threshold | fs2 > tf_threshold) &   pvec[ id * pvec_max + ipos] != 4 ){
            output[ 10 * ichunk + icol ] = fs34;
            output[ 11 * ichunk + icol ] = fs44;
         } 


      }
 

   }

}
 

  void tfbs_cuda(  char *pvec,   // promoter
                   int *pvec_len,
                   int pvec_max,
                   float *pwm, 
                   int pwm_len, 
                   float tf_threshold, 
                   float *bg, 
                   int N, 
                   float *output)  {// output matrix


   
     char *d_pvec;
     int *d_pvec_len;
     float *d_output;
     float *d_pwm;
     float *d_bg;

   

	//printf("Allocating GPU memory for pvec_len\n");
	gpuErrchk( hipMalloc( (void**)&d_pvec_len, N * sizeof(  int)) );
	//printf("Copying GPU memory for pvec_len\n");
	gpuErrchk( hipMemcpy( d_pvec_len, pvec_len, N * sizeof(  int), hipMemcpyHostToDevice ) ); 

	//printf("Allocating GPU memory for pvec\n");
	gpuErrchk( hipMalloc( (void**) &d_pvec, N * pvec_max * sizeof(  char)) );
	//printf("Copying GPU memory for pvec\n");
        gpuErrchk( hipMemcpy(     d_pvec, pvec, N * pvec_max * sizeof(  char), hipMemcpyHostToDevice ) ); 

	//printf("Allocating GPU memory for pwm\n");
 	gpuErrchk( hipMalloc( (void**)&d_pwm, 4 * pwm_len * sizeof(float)) );
	//printf("Copying GPU memory for pwm\n");
	gpuErrchk( hipMemcpy(     d_pwm, pwm, 4 * pwm_len * sizeof(float), hipMemcpyHostToDevice ) ); 


	//printf("Allocating GPU memory for bg\n");
 	gpuErrchk( hipMalloc( (void**)&d_bg, 4 *  sizeof(float)) );
	//printf("Copying GPU memory for bg\n");
	gpuErrchk( hipMemcpy(      d_bg, bg, 4 *  sizeof(float), hipMemcpyHostToDevice ) ); 


	//printf("Allocating GPU memory for result\n");
	gpuErrchk( hipMalloc( (void**)&d_output, N * 12 * (pvec_max -58 ) * sizeof(float) ) ); 
	//printf("Before kernel\n");
	
	dim3 dimBlock( 32, 1 );
	dim3 dimGrid( N/32 + 1,  1 );
	//printf("Calling CUDA kernel\n");
	tfbs_kernel<<<dimGrid,dimBlock>>>(N, pvec_max,
                                          d_pvec, d_pvec_len, 
                                          d_pwm,  pwm_len, 
                                          tf_threshold, d_bg, 
                                          d_output);
	gpuErrchk( hipPeekAtLastError() );

	//cudaDeviceSynchronize();
	//printf("After CUDA kernel\n");

	gpuErrchk( hipMemcpy( output, d_output, N * 12 * (pvec_max -58 ) * sizeof(float), hipMemcpyDeviceToHost ) ); 
	//printf("Free GPU memory \n");

	hipFree( d_pvec );
	hipFree( d_pvec_len );
	hipFree( d_output );

	hipFree( d_pwm );
	hipFree( d_bg );


   return;
  }



int main( int argc, char *argv[] ){

   char filename[1024];
   int taskID;
   float bg[] = {0.25, 0.25, 0.25, 0.25};
   FILE *ifp, *ofp;
   char in_line[MAX_LINE_LEN];
   char *token;
   int chrom[PROMOTER_LEN];
   int istart[PROMOTER_LEN];
   int i, j, jj, i1, i2, j1, j2, icurr;
   float score_threshold[SCORE_LEN];
   char **score_files;
   float pwm[30 * 4];
   int pwm_length;
   int any0;
   float sum;
   char *pnvec;
   int *pnvec_len; 
   float *result;
   int ires;
 

   /* command line processing */
   taskID = atoi(argv[1]); 
   printf("Input file name: %s\n", argv[2]);
   printf("tf.info file name: %s\n", argv[3]);
   printf("pwm directory: %s\n", argv[4]);
   sprintf(filename,"./%s/%s_%03d.txt", argv[5], argv[6], taskID);
   printf("Output file name: %s\n", filename);


   /* allocate memory to hold char arrays */
   //sequence = (char **) malloc( PROMOTER_LEN * sizeof(char *) );
   score_files = (char **) malloc( SCORE_LEN * sizeof(char *) );
   result = (float *) malloc( Nchunk * 12 * ( PNVEC_MAX - 58 ) * sizeof(float) );

   pnvec_len = ( int * ) malloc ( PROMOTER_LEN * sizeof( int ) );
   if ( pnvec_len == 0 ) {
      fprintf( stderr, "ERROR allocating pnvec_len: Out of memory\n");
      exit(1);
   }

   //printf("Allocating memory for pnvec\n");
   pnvec = ( char * ) malloc ( PROMOTER_LEN * PNVEC_MAX * sizeof( char) );
   if ( pnvec == 0 ) {
      fprintf( stderr, "ERROR allocating pnvec: Out of memory\n");
      exit(2);
   }

   printf("Reading Input files\n");

   /* read input file line by line  (only 1st, 2nd and 8th columns) */
   ifp = fopen(argv[2], "r");
   fgets(in_line, MAX_LINE_LEN, ifp); //skip header line
   i=0;
   while( fgets(in_line, MAX_LINE_LEN, ifp )!= NULL ){

      token = strtok( in_line, DELIM);
      sscanf( token,"chr%d", &(chrom[i])); 

      token = strtok( NULL, DELIM);
      sscanf( token,"%d", istart + i ); 

      token = strtok( NULL, DELIM);
      token = strtok( NULL, DELIM);
      token = strtok( NULL, DELIM);
      token = strtok( NULL, DELIM);
      token = strtok( NULL, DELIM);
      token = strtok( NULL, DELIM);

      //if(i > 14655) printf(" start processing letter\n");
      pnvec_len[i] = strlen(token);
      for (j = 0; j < pnvec_len[i] ; j++) {
         //if (i == 14660)printf("%c",token[j]);
         switch( token[j] ){
         case 'A':
            pnvec[ i * PNVEC_MAX + j ]=1;
            break;
         case 'a':
            pnvec[ i * PNVEC_MAX + j ]=1;
            break;
         case 'C':
            pnvec[ i * PNVEC_MAX + j ]=2;
            break;
         case 'c':
            pnvec[ i * PNVEC_MAX + j ]=2;
            break;
         case 'G':
            pnvec[ i * PNVEC_MAX + j ]=3;
            break;
         case 'g':
            pnvec[ i * PNVEC_MAX + j ]=3;
            break;
         case 'T':
            pnvec[ i * PNVEC_MAX + j ]=4;
            break;
         case 't':
            pnvec[ i * PNVEC_MAX + j ]=4;
            break;
         default:
            pnvec[ i * PNVEC_MAX + j ]=0;
            break;
         }
      }

      i++;
      //if ( i > 14600) { printf("i=%d\n",i);}
   }
   fclose(ifp);
   printf(" Read %d lines from the input file\n", i); 


   /* Read tf.info file */
   ifp = fopen(argv[3], "r");
   i=0;
   while( fgets(in_line, MAX_LINE_LEN, ifp ) ){

      token = strtok( in_line, DELIM);
      score_files[i] = (char *) malloc ( (strlen(token) + 1 ) * sizeof(char ));
      strcpy( score_files[i], token );

      token = strtok( NULL, DELIM);
      score_threshold[i] = atof(token);

      i++;


   }

   fclose(ifp);
   printf(" Read %d lines from %s file\n", i, argv[3]); 


   /* process chunks */
   i1 = (taskID - 1) * 100 + 1;  // was 10 originally
   i2 = taskID * 100;
   if ( i2 > SCORE_LEN ) i2 = SCORE_LEN;

   /* open output file */
   ofp = fopen(filename,"w");

   if (ofp == NULL) {
      fprintf( stderr, " Can't open output file\n");
      exit(3);
   }


   for ( icurr = i1; icurr <= i2; icurr++){
      printf(" icurr =%d\n", icurr);

      sprintf( filename, "./%s/%s\0", argv[4], score_files[icurr-1] );
      ifp = fopen( filename , "r");
      fgets(in_line, MAX_LINE_LEN, ifp ); // skip first line
      i = 0;
      any0 = 0;
      while( fgets(in_line, MAX_LINE_LEN, ifp ) ){

         token = strtok( in_line, DELIM); //skip first value

         token = strtok( NULL, DELIM);
         pwm[i*4 + 0] = atof(token); 
         if ( !strcmp(token, "0.0") ) any0=1;

         token = strtok( NULL, DELIM);
         pwm[i*4 + 1] = atof(token); 
         if ( !strcmp(token, "0.0") ) any0=1;

         token = strtok( NULL, DELIM);
         pwm[i*4 + 2] = atof(token); 
         if ( !strcmp(token, "0.0") ) any0=1;

         token = strtok( NULL, DELIM);
         pwm[i*4 + 3] = atof(token); 
         if ( !strcmp(token, "0.0\n") ) any0=1;
         
         i++;
      }

      fclose(ifp);
      pwm_length = i;
      printf(" Read %d lines from %s file\n", i, score_files[icurr-1]); 

      /* part of create_pwm function */
      if ( any0 ) {
         for ( j = 0; j < i; j++ ){
           sum = pwm[ j*4 + 0] + pwm[ j*4 + 1] + pwm[ j*4 + 2] + pwm[ j*4 + 3] + 0.001 * 4;
           pwm[ j*4 + 0] = (pwm[ j*4 + 0] + 0.001)/sum;
           pwm[ j*4 + 1] = (pwm[ j*4 + 1] + 0.001)/sum;
           pwm[ j*4 + 2] = (pwm[ j*4 + 2] + 0.001)/sum;
           pwm[ j*4 + 3] = (pwm[ j*4 + 3] + 0.001)/sum;

         }
      }


      /* inner loop in R*/
      for ( j = 1; j < 12; j++ ){

         j1 = (j - 1) * Nchunk + 1;
         j2 = j * Nchunk;
         if ( j2 > PROMOTER_LEN )j2 = PROMOTER_LEN;
         int n = j2 - j1 + 1;

         printf(" j = %d through %d; threshold = %f\n", j1, j2, score_threshold[icurr - 1]);
         tfbs_cuda (pnvec + (j1 -1) * PNVEC_MAX, 
                    pnvec_len + j1 -1, 
                    PNVEC_MAX, 
                    pwm, 
                    pwm_length, 
                    score_threshold[icurr - 1], 
                    bg, 
                    n,  
                    result);

         fflush(stdout);

         /* save result in the output file */
         for (i = 0; i < n * ( PNVEC_MAX - 58 ); i++){
            ires = (int) result [ i ];
            int in = i/( PNVEC_MAX - 58 );
            //printf("%d ",i);
            if (ires > 0 || (score_threshold[ icurr - 1] < result[ i + ( PNVEC_MAX - 58 ) * 2 * n] )|| (score_threshold[icurr - 1] < result[i + (PNVEC_MAX - 58 ) * 3 * n])) {
               fprintf(ofp,"%d ", chrom [ in ]);
               //unsigned int ipos = result [ i +  ( PNVEC_MAX - 58 ) * n * 1] + istart[ j1 - 1 + i/( PNVEC_MAX - 58 ) ]; 
               unsigned int ipos = 30 + i%( PNVEC_MAX - 58 ) + istart[ j1 - 1 + in ]; 
               fprintf(ofp,"%d %d ", ipos, ipos + 1);
               fprintf(ofp,"%d %d ", j1 + in,  icurr);
               fprintf(ofp,"%d ", ires );
               // use %f.3 for printing results with 3 digits
               for ( jj = 2; jj < 11; jj++ )fprintf(ofp,"%.3f ", result [ i +  ( PNVEC_MAX - 58 ) * jj * n] );
               fprintf(ofp,"%.3f\n", result [ i + ( PNVEC_MAX - 58 ) * 11 * n] );
            }
         } 

      } // end of j loop


   } // end of icurr loop
 
   fclose(ofp);

   exit(0);
}
